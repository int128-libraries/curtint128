#include "hip/hip_runtime.h"
#include <iostream>
#include <stdint.h>
#include <math.h>
#include <string>
#include <hiprand.h>

#include <CUDASieve/cudasieve.hpp>
#include <CUDASieve/host.hpp>

#include "cuda_uint128.h"
#include "cuda_uint128_primitives.cuh"
#include "utils.h"

uint64_t * generateUniform64(uint64_t num);

__global__
void atimesbequalsc(uint64_t * a, uint64_t * b, uint128_t * c);
__global__
void squarerootc(uint128_t * c, uint64_t * a);
__global__
void sqrt_test(uint64_t * a, volatile uint64_t * errors);
__global__
void div_test(uint64_t * a, volatile uint64_t * errors);


int main(int argc, char * argv[])
{
  uint128_t x = (uint128_t) 1 << 120;

  if(argc == 2)
    x = string_to_u128((std::string)argv[1]);

  #pragma omp parallel for
  for(uint64_t v = 2; v < 1u << 30; v++){
    uint64_t r;
    uint128_t y = uint128_t::div128to128(x, v, &r);
    uint128_t z = mul128(y, v) + r;

    if(z != x) std::cout << z << std::endl;
  }

  // uint64_t * d64 = generateUniform64(1u<<26);
  // volatile uint64_t * h_errors, * d_errors;
  // hipHostAlloc((void **)&h_errors, sizeof(uint64_t), hipHostMallocMapped);
  // hipHostGetDevicePointer((uint64_t **)&d_errors, (uint64_t *)h_errors, 0);
  //
  // *h_errors = 0;
  //
  // KernelTime timer;
  //
  // timer.start();
  //
  // div_test<<<65536, 256>>>(d64, d_errors);
  //
  // hipDeviceSynchronize();
  // timer.stop();
  // timer.displayTime();
  //
  // std::cout << *h_errors << " errors " << std::endl;

  return 0;
}

uint64_t * generateUniform64(uint64_t num)
{
  uint64_t * d_r;
  hiprandGenerator_t gen;

  hipMalloc(&d_r, num * sizeof(uint64_t));

  hiprandCreateGenerator(&gen, HIPRAND_RNG_QUASI_SOBOL64);
  hiprandSetPseudoRandomGeneratorSeed(gen, 1278459ull);
  hiprandGenerateLongLong(gen, (unsigned long long *)d_r, num);

  return d_r;
}

__global__
void atimesbequalsc(uint64_t * a, uint64_t * b, uint128_t * c)
{
  uint32_t tidx = threadIdx.x + blockDim.x * blockIdx.x;
  c[tidx] = mul128(a[tidx], b[tidx]);
}

__global__
void squarerootc(uint128_t * c, uint64_t * a)
{
  uint32_t tidx = threadIdx.x + blockDim.x * blockIdx.x;
  a[tidx] = _isqrt(c[tidx]);
  if(mul128(a[tidx], a[tidx]) > c[tidx] || mul128((a[tidx] + 1), (a[tidx] + 1)) <= c[tidx])
    printf("%llu  %f  %llu\n", a[tidx], u128_to_float(c[tidx]), c[tidx].hi);
}

__global__
void sqrt_test(uint64_t * a, volatile uint64_t * errors)
{
  __shared__ uint64_t s_a[1024];

  #pragma unroll
  for(uint16_t i = 0; i < 4; i++){
    s_a[threadIdx.x + i * blockDim.x] = a[threadIdx.x + i * blockDim.x + 1024*blockIdx.x];
  }
  __syncthreads();

  uint128_t x;
  #pragma unroll
  for(uint16_t i = 0; i < 4; i++){
    x.lo = s_a[threadIdx.x + i * blockDim.x];
    #pragma unroll
    for(uint16_t i = 0; i < 1024; i++){
      x.hi = s_a[i] >> 4;
      uint64_t y = _isqrt(x);
      if(mul128(y,y) > x || mul128(y + 1, y + 1) <= x){
        atomicAdd((unsigned long long *)errors, 1ull);
        printf("%llu %llu %llu\n", x.hi, x.lo, y);
      }
    }
  }
}

__global__
void div_test(uint64_t * a, volatile uint64_t * errors)
{
  __shared__ uint64_t s_a[1024];

  #pragma unroll
  for(uint16_t i = 0; i < 4; i++){
    s_a[threadIdx.x + i * blockDim.x] = a[threadIdx.x + i * blockDim.x + 1024*blockIdx.x];
  }
  __syncthreads();

  uint128_t x, y;
  uint64_t v, r;
  #pragma unroll
  for(uint16_t i = 0; i < 4; i++){
    x.lo = s_a[threadIdx.x + i * blockDim.x];
    #pragma unroll
    for(uint16_t i = 0; i < 1024; i++){
      x.hi = s_a[i] >> 4;
      v = s_a[(i + 1 )& 1023] >> (x.hi & 31);
      y = div128to128(x,v,&r);
      y = add128(mul128(y, v), r);
      uint64_t y = _isqrt(x);
      // if(y != x){
      //   atomicAdd((unsigned long long *)errors, 1ull);
      // }
    }
  }
}
